#include "hip/hip_runtime.h"
#define WIN32
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

/////////////////////////////////////////////

void startAndHandleCmdLine(int argc, char** argv, int& devID)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char**)argv, "help") ||
        checkCmdLineFlag(argc, (const char**)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    if (checkCmdLineFlag(argc, (const char**)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char**)argv, "device");
        hipSetDevice(devID);
    }
}

void handleErrors(int devID, hipDeviceProp_t deviceProp)
{
    hipError_t error;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
}

void cmdLineDimUpdate(int argc, char** argv, dim3& dimsA, dim3& dimsB)
{
    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char**)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char**)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char**)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char**)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char**)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char**)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char**)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char**)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
            dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);
}

void allocateDeviceMemory(float*& d_A, float*& d_B, float*& d_C,
    unsigned int mem_size_A, unsigned int mem_size_B, unsigned int mem_size_C)
{
    hipError_t error;

    error = hipMalloc((void**)&d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void**)&d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void**)&d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
}

void copyHostMemoryToDevice(float*& d_A, float*& d_B, float* h_A, float* h_B,
    unsigned int mem_size_A, unsigned int mem_size_B)
{
    hipError_t error;
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
}

void constantInit(float* data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/////////////////////////////////////////////

template <int BLOCK_SIZE> __global__
void matrixMulCUDA(float* C, float* A, float* B, int N)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    for (int k = 0; k < N; k++) {
        // Accumulate results for a single element
        Csub += A[row * N + k] * B[k * N + col];
    }

    C[row * N + col] = Csub;
}

int matrixMultiply(int argc, char** argv, int block_size, dim3& dimsA, dim3& dimsB)
{
    // Allocate host memory for matrices A, B and C
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float* h_C = (float*)malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float* d_A, * d_B, * d_C;
    allocateDeviceMemory(d_A, d_B, d_C, mem_size_A, mem_size_B, mem_size_C);

    // Copy host memory to device
    copyHostMemoryToDevice(d_A, d_B, h_A, h_B, mem_size_A, mem_size_B);

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    /*
    if (block_size == 16)
    {
        matrixMulCUDA<16> << < grid, threads >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
        matrixMulCUDA<32> << < grid, threads >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");
    */

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    hipError_t error;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
        if (block_size == 16)
        {
            matrixMulCUDA<16> << < grid, threads >> > (d_C, d_A, d_B, dimsA.x);
        }
        else
        {
            matrixMulCUDA<32> << < grid, threads >> > (d_C, d_A, d_B, dimsA.x);
        }
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        if (fabs(h_C[i] - (dimsA.x * valB)) > 1e-5)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > 1e-5\n", i, h_C[i], dimsA.x * valB);
            correct = false;
        }
    }

    printf("%s\n", correct ? "OK" : "FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNote: For peak performance, please refer to the matrixMulCUBLAS example.\n");

    hipDeviceReset();

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

/////////////////////////////////////////////

int main(int argc, char** argv)
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;
    startAndHandleCmdLine(argc, argv, devID);

    hipDeviceProp_t deviceProp;
    handleErrors(devID, deviceProp);

    // Use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;

    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 2 * block_size, 5 * 2 * block_size, 1);
    cmdLineDimUpdate(argc, argv, dimsA, dimsB);

    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}
